#include "hip/hip_runtime.h"
#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <thrust/device_vector.h>
#include <>
#include <hipblas.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <stdio.h>
#include <time.h>

// C-style indexing
int ci(int row, int column, int nColumns)
{
	return row * nColumns + column;
}

float* h_A;
float* h_B;
float* h_C;

int rowA = 10;
int colA = 8;

int rowB = colA;
int colB = 2;

int rowC = rowA;
int colC = colB;

hipblasHandle_t handle;
hipblasStatus_t status;

void device_info();
void gpu_random_init(float* A, int rows, int cols);

//void copy_from_gpu_to_cpu(float* A, float* B);

void free_all_memory();

void gpu_matrix_product(const float* A, const float* B, float* C, const int m, const int k, const int n);
void cpu_matrix_product();

void matrix_print(float* A, int rows, int cols);

int main()
{
	device_info();

	// allocate three device_vectors with row*col elements
	thrust::device_vector<float> d_A(rowA * colA);
	thrust::device_vector<float> d_B(rowB * colB);
	thrust::device_vector<float> d_C(rowC * colC);

//	gpu_random_init(raw_pointer_cast(&d_A[0]), rowA, colA);
	//gpu_random_init(raw_pointer_cast(&d_B[0]), rowB, colB);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	//gpu_matrix_product(raw_pointer_cast(&d_A[0]), raw_pointer_cast(&d_B[0]), raw_pointer_cast(&d_C[0]), rowA, colA, colB);
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("CUBLAS time : %f ms\n", milliseconds);
	
	//matrix_print(raw_pointer_cast(&d_A[0]), rowA, colA);
	//matrix_print(raw_pointer_cast(&d_B[0]), rowB, colB);
	//matrix_print(raw_pointer_cast(&d_C[0]), rowC, colC);

//	copy_from_gpu_to_cpu(raw_pointer_cast(&d_A[0]), raw_pointer_cast(&d_B[0]));
	
	status = hipblasDestroy(handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
		printf("shutdown error %s \n", status);

	return 0;
}


void device_info()
{
	int kb = 1024;
	int mb = kb * kb;

	int GPU_N;
	hipGetDeviceCount(&GPU_N);
	printf("Device count: %d\n", GPU_N);

	for (int i = 0; i < GPU_N; i++)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		printf("PCI Bus id: %d\n", props.pciBusID);

		hipGetDeviceProperties(&props, i);
		printf("Device %i: %s: %i.%i\n", i, props.name, props.major, props.minor);
		printf("Global memory: %i mb\n", props.totalGlobalMem / mb);
		printf("Shared memory: %i kb\n", props.sharedMemPerBlock / kb);
		printf("Constant memory:  %i kb\n", props.totalConstMem / kb);
		printf("Block registers: %i\n", props.regsPerBlock);
		printf("Warp size: %i\n", props.warpSize);
		printf("Threads per block: %i\n", props.maxThreadsPerBlock);
		printf("Max block dimensions: [ %i, %i, %i]\n", props.maxThreadsDim[0], props.maxThreadsDim[1],
		       props.maxThreadsDim[2]);
		printf("Max grid dimensions:  [ %i, %i, %i]\n", props.maxGridSize[0], props.maxGridSize[1], props.maxGridSize[2]);
	}
}

//void copy_from_gpu_to_cpu(float* A, float* B)
//{
//	h_A = (float *)malloc(rowA * colA * sizeof(float)); 
//	h_B = (float *)malloc(rowB * colB * sizeof(float)); 
//	h_C = (float *)malloc(rowC * colC * sizeof(float));
//
//	hipMemcpy(h_A, A, rowA * colA * sizeof(float), hipMemcpyDeviceToHost);
//	hipMemcpy(h_B, B, rowB * colB * sizeof(float), hipMemcpyDeviceToHost);
//}

// C(m,n) = A(m,k) * B(k,n)
void gpu_matrix_product(const float* A, const float* B, float* C, const int m, const int k, const int n)
{
	// Initialize CUBLAS 
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS)
		printf("CUBLAS initialization error with message %s\n", status);

	float alpha = 1.0f;
	float beta = 0.0f;

	//C = alpha*A*B + beta * C
	status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colB, rowA, colA, &alpha, B, colB, A, colA, &beta, C, colC);

	if (status != HIPBLAS_STATUS_SUCCESS)
		printf("Kernel execution error with message %s\n", status);
}

void cpu_matrix_product()
{

}

void gpu_random_init(float* A, int rows, int cols)
{
	// Create a pseudo-random number generator
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

	// Set the seed for the random number generator using the system clock
	hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long)clock());

	// Fill the array with random numbers on the device
	size_t n = rows * cols;
	hiprandGenerateUniform(gen, A, n);
	hiprandDestroyGenerator(gen); /* Cleanup */
}

void matrix_print(float* A, int rows, int cols)
{
	for (size_t i = 0; i < rows; i++)
	{
		for (size_t j = 0; j < cols; j++)
		{
			std::cout << A[ci(i, j, cols)] << " ";
		}
		printf("\n");
	}
}
